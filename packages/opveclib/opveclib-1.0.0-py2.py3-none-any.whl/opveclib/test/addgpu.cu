#include "hip/hip_runtime.h"
/* Copyright 2016 Hewlett Packard Enterprise Development LP

 Licensed under the Apache License, Version 2.0 (the "License"); you may not use this file except in compliance with
 the License. You may obtain a copy of the License at

 http://www.apache.org/licenses/LICENSE-2.0
 Unless required by applicable law or agreed to in writing, software distributed under the License is distributed
 on an "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the License for
 the specific language governing permissions and limitations under the License.*/

#include "dynamiclibop.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <memory>
#include <assert.h>

// turn off c++ name mangling
#define ADDGPU_EXPORT extern "C"

// GPU functions to be called by the TF dynamic_lib_addgpu_test.py
// code must be compilable by nvcc

// cuda kernels

__global__ void Add2GPUKernel(const float *in0, const float *in1,  float* out, int size) {
  const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  const int total_thread_count = gridDim.x * blockDim.x;

  int offset = thread_id;

  while (offset < size) {
    out[offset] = in0[offset] + in1[offset] + 1.0;
    offset += total_thread_count;
  }
}

__global__ void Add3GPUKernel(const float *in0, const double *in1,  const float *in2, float* out, int size) {
  const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  const int total_thread_count = gridDim.x * blockDim.x;

  int offset = thread_id;

  while (offset < size) {
    out[offset] = in0[offset] + in1[offset] + in2[offset] + 1.0;
    offset += total_thread_count;
  }
}

__global__ void SumSqGPUKernel(const float *in0, const double *in1,  float* out0, float *out1, int size) {
  const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  const int total_thread_count = gridDim.x * blockDim.x;

  int offset = thread_id;

  while (offset < size) {
    out0[offset] = in0[offset] + in1[offset] + 1.0;
    out1[offset] = out0[offset] * out0[offset] + 1.0;
    offset += total_thread_count;
  }
}

// dynamic library operators

ADDGPU_EXPORT
void add2float(std::vector<std::shared_ptr<const InputParameter>> inputs,
		      std::vector<std::shared_ptr<OutputParameter>> outputs, hipStream_t stream,
		      uint16_t threads_per_block, uint16_t *err) {
	if (inputs.size() != 2) { *err = 1; return; }
	if (outputs.size() != 1) { *err = 1; return; }

	float *out = outputs[0]->get<float>();
	const float *in0 = inputs[0]->get<float>();
	const float *in1 = inputs[1]->get<float>();
	int64_t len = inputs[0]->length();
	uint32_t num_blocks = len / threads_per_block;
	if(len % threads_per_block > 0) num_blocks += 1;

	std::cout << "*** launching Add2GPUKernel ***" << std::endl;
	Add2GPUKernel<<<num_blocks, threads_per_block, 0, stream>>>(in0, in1, out, len);
}

ADDGPU_EXPORT
void addFloatDoubleFloat(std::vector<std::shared_ptr<const InputParameter>> inputs,
		      std::vector<std::shared_ptr<OutputParameter>> outputs, hipStream_t stream,
		      uint16_t threads_per_block, uint16_t *err) {
	if (inputs.size() != 3) { *err = 1; return; }
	if (outputs.size() != 1) { *err = 1; return; }


	float *out = outputs[0]->get<float>();
	const float *in0 = inputs[0]->get<float>();
	const double *in1 = inputs[1]->get<double>();
	const float *in2 = inputs[2]->get<float>();
	int64_t len = inputs[0]->length();
	uint32_t num_blocks = len / threads_per_block;
	if(len % threads_per_block > 0) num_blocks += 1;

	std::cout << "*** launching Add3GPUKernel ***" << std::endl;
	Add3GPUKernel<<<num_blocks, threads_per_block, 0, stream>>>(in0, in1, in2, out, len);
}

ADDGPU_EXPORT
void sumAndSq(std::vector<std::shared_ptr<const InputParameter>> inputs,
		      std::vector<std::shared_ptr<OutputParameter>> outputs, hipStream_t stream,
		      uint16_t threads_per_block, uint16_t *err) {
	if (inputs.size() != 2) { *err = 1; return; }
	if (outputs.size() != 2) { *err = 1; return; }

	float *out0 = outputs[0]->get<float>();
	float *out1 = outputs[1]->get<float>();
	const float *in0 = inputs[0]->get<float>();
	const double *in1 = inputs[1]->get<double>();
	int64_t len = inputs[0]->length();
	uint32_t num_blocks = len / threads_per_block;
	if(len % threads_per_block > 0) num_blocks += 1;

	std::cout << "*** launching SumSqGPUKernel ***" << std::endl;
	SumSqGPUKernel<<<num_blocks, threads_per_block, 0, stream>>>(in0, in1, out0, out1, len);
}




